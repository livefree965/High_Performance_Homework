#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <fstream>
using namespace std;
template <int BLOCK_SIZE> __global__ void
MatMul_CUDA(float *A, float *B, float *C, int mat_a_col, int mat_b_col)
{
	
	int block_x = blockIdx.x, block_y = blockIdx.y; // Block index
	int thread_x = threadIdx.x, thread_y = threadIdx.y; // Thread index

	
	int mat_a_begin = mat_a_col * BLOCK_SIZE * block_y;// Index of the first sub-matrix of A processed by the block
	int mat_a_end = mat_a_begin + mat_a_col - 1;// Index of the last sub-matrix of A processed by the block
	int mat_a_step = BLOCK_SIZE;// Step size used to iterate through the sub-matrices of A

	
	int mat_b_begin = BLOCK_SIZE * block_x; // Index of the first sub-matrix of B processed by the block
	int mat_b_step = BLOCK_SIZE * mat_b_col; // Step size used to iterate through the sub-matrices of B

	float c_sub_res = 0; // c_sub_res is used to store the element of the block sub-matrix

	// Loop  all the sub-matrices of A and B
	for (int a = mat_a_begin, b = mat_b_begin; a <= mat_a_end; a += mat_a_step, b += mat_b_step)
	{

		// A_sub,B_sub used to store the sub-matrix of A and B
		__shared__ float A_sub[BLOCK_SIZE][BLOCK_SIZE],B_sub[BLOCK_SIZE][BLOCK_SIZE];
		A_sub[thread_y][thread_x] = A[a + mat_a_col * thread_y + thread_x]; // Load A,B sub-matrices from device memory to share memory
		B_sub[thread_y][thread_x] = B[b + mat_b_col * thread_y + thread_x]; // Each thread loadsone element of each matrix

		__syncthreads(); // Make sure the matrix are loaded
		
		// Multiply the two matrix
		for (int k = 0; k < BLOCK_SIZE; ++k)
			c_sub_res += A_sub[thread_y][k] * B_sub[k][thread_x];

		// Synchronize to make sure that the preceding computation is finish
		__syncthreads();
	}

	// Write the block sub-matrix to device memory, each thread writes one element
	int c = mat_b_col * BLOCK_SIZE * block_y + BLOCK_SIZE * block_x;
	C[c + mat_b_col * thread_y + thread_x] = c_sub_res;
}

void show_mat(float**src, int col, int row) {
	/*
		src:address of the begin of matrix
		col:col size
		row:row size
	*/
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			printf("%f ", (*src)[i*col + j]);
		}
		printf("\n");
	}
}

void save_mat(float**src, int col, int row) {
	/*
		src:address of the begin of matrix
		col:col size
		row:row size
	*/
	ofstream outfile("mat_c.txt");
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			outfile << i << " " << j << " " << (*src)[i*col + j] << endl;
		}
	}
	outfile.close();
	printf("save finish");
}
void InitMatA(float **A, int col, int row) {
	/*
		A:address of begin of matrix
		col:col size
		row:row size
	*/
	*A = (float*)malloc(col*row * sizeof(float));
	for (int i = 0; i < row; i++) {
		for (int j = 0; j < col; j++) {
			(*A)[i*col + j] = i - 0.1*j + 1;
		}
	}
}
void InitMatB(float **B, int col, int row) {
	/*
		B:address of begin of matrix
		col:col size
		row:row size
	*/
	*B = (float*)malloc(col*row * sizeof(float));
	for (int i = 0; i < row; i++) {
		for (int j = 0; j < col; j++) {
			(*B)[i*col + j] = 0.2*j - 0.1*i;
		}
	}
}

void MatMul_CPU(float *A, float *B, float **C, int left_col, int left_row, int right_col ) {
	/*
		Mat a multi b serial version
		A: left matrix
		B: right matrix
		C: result matrix
		left_col: left matrix column size
		right_col: like above
		left row: like above
	*/
	clock_t beg, end;
	printf("Computing in CPU...\n");
	beg = clock();
	*C = (float*)malloc(left_row*right_col * sizeof(float));
	float *tmp = *C;
	float sum;
	int mat_a_index_base;
	for (int i = 0; i < left_row; i++) {
		for (int j = 0; j < right_col; j++) {
			sum = 0.0f;
			mat_a_index_base = i * left_col;
			for (int k = 0; k < left_col; k++) {
				sum += A[mat_a_index_base + k] * B[k*right_col + j];
			}
			tmp[i*right_col + j] = sum;
		}
	}
	end = clock();
	printf("Time= %.3f msec\n", (double)(end - beg)*1000 / CLOCKS_PER_SEC);
}

void MatMul_GPU(float *A, float *B, float **C, int block_size, dim3 &dims_A, dim3 &dims_B) {
	// Calculate malloc size for matrix A, B, C
	unsigned int size_A = dims_A.x * dims_A.y; //Size of matrix A
	unsigned int mem_size_A = sizeof(float) * size_A; //Memory size of matrix A
	unsigned int size_B = dims_B.x * dims_B.y;
	unsigned int mem_size_B = sizeof(float) * size_B;
	dim3 dimsC(dims_B.x, dims_A.y, 1);
	unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
	*C = (float*)malloc(mem_size_C);

	//Above allocate memory for host,now malloc for device
	float *CUDA_A, *CUDA_B, *CUDA_C;

	hipError_t error_A, error_B, error_C;
	error_A = hipMalloc((void **)&CUDA_A, mem_size_A);
	error_B = hipMalloc((void **)&CUDA_B, mem_size_B);
	error_C = hipMalloc((void **)&CUDA_C, mem_size_C);

	//Allocate memory to store matrix in GPU

	if (error_A != hipSuccess || error_B != hipSuccess || error_C != hipSuccess)
	{
		printf("hipMalloc failed");
		exit(EXIT_FAILURE);
	}

	//Copy matrix from A to CUDA_A
	error_A = hipMemcpy(CUDA_A, A, mem_size_A, hipMemcpyHostToDevice);
	error_B = hipMemcpy(CUDA_B, B, mem_size_B, hipMemcpyHostToDevice);
	
	if (error_A != hipSuccess || error_B != hipSuccess)
	{
		printf("hipMemcpy failed");
		exit(EXIT_FAILURE);
	}

	// To one thread,solve one ceil in matrix c
	//All thread num is matrix c size
	dim3 blocks(block_size, block_size);
	dim3 grid(dims_B.x / blocks.x, dims_A.y / blocks.y);

	// Begin compute
	printf("Computing in GPU...\n");

	//Warm up
	MatMul_CUDA<16> << < grid, blocks >> > (CUDA_A, CUDA_B, CUDA_C, dims_A.x, dims_B.x);
	printf("Warmup Finish\n");

	//Synchronize the gpu
	hipDeviceSynchronize();

	//Record the time
	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventRecord(begin, NULL);

	MatMul_CUDA<16> << < grid, blocks >> > (CUDA_A, CUDA_B, CUDA_C, dims_A.x, dims_B.x);

	printf("Finish\n");
	hipEventCreate(&end);
	hipEventRecord(end, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(end);

	float time_total = 0.0f;
	hipEventElapsedTime(&time_total, begin, end);

	// Compute and print the performance
	printf(
		"Time= %.3f msec \n",
		time_total);

	// Copy result from gpu
	error_C = hipMemcpy(*C, CUDA_C, mem_size_C, hipMemcpyDeviceToHost);

	if (error_C != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n",
			hipGetErrorString(error_C), error_C, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Free memory
	hipFree(CUDA_A);
	hipFree(CUDA_B);
	hipFree(CUDA_C);

	// Use driver to clean up all state
	hipDeviceReset();
}

bool Check_res(float *serial_result, float *cuda_result, int length) {
	bool correct = true;
	// test relative error by (<x, y>_cpu - <x,y>_gpu)  < eps
	double eps = 1.e-3; // machine zero
	double percent = 0.0f;
	for (int i = 0; i < length; i++)
	{
		double abs_err = fabs(cuda_result[i] - serial_result[i]);
		percent = abs_err / serial_result[i];
		if (percent > eps)
		{
			printf("Wrong at [%d] cpu=%.8f, gpu=%.8f percent=%.2f \n",
				i, serial_result[i], cuda_result[i], percent * 100);
			correct = false;
		}
	}
	printf("%s\n", correct ? "Result PASS" : "Result FAIL");
	return correct;
}
/**
 * Program main
 */
int main(int argc, char **argv)
{
	// Use Geforce 960M
	int devID = 0;

	// input five parameters
	int block_size = strtol(argv[1], NULL, 10);
	int mat_a_col = strtol(argv[2], NULL, 10);
	int mat_a_row = strtol(argv[3], NULL, 10);
	int mat_b_col = strtol(argv[4], NULL, 10);
	int mat_b_row = strtol(argv[5], NULL, 10);

	//see if we calculate right
	bool res;


	float *A, *B, *C1, *C2;
	//C1: calculate by gpu
	//C2: calculate by cpu

	hipError_t error;
	error = hipSetDevice(devID);

	if (error != hipSuccess)
	{
		printf("hipSetDevice wrong");
	}

	dim3 dims_A(mat_a_col, mat_a_row, 1);
	dim3 dims_B(mat_b_col, mat_b_row, 1);

	//to partition,we require that
	if (mat_a_col % block_size != 0 || mat_a_row % block_size != 0 ||
		mat_b_col % block_size != 0 || mat_b_row % block_size != 0) {
		printf("Dimension size must be dividable by block size!\n");
		exit(1);
	}

	//the condition to satisfiy a*b
	if (dims_A.x != dims_B.y)
	{
		printf("Outer matrix dimensions must be equal");
		exit(1);
	}

	// Init matrix
	InitMatA(&A, mat_a_col, mat_a_row);
	InitMatB(&B, mat_b_col, mat_b_row);
	
	
	//show_mat(&A, mat_a_w, mat_a_h);
	//printf("\n");
	//show_mat(&B, mat_b_w, mat_b_h);
	
	//GPU calculate
	MatMul_GPU(A, B, &C1, block_size, dims_A, dims_B);
	
	//CPU calculate
	printf("\n");
	MatMul_CPU(A, B, &C2, mat_a_col, mat_a_row, mat_b_col);
	
	//compare if the res is the same
	res = Check_res(C2, C1, mat_a_row * mat_b_col);
	
	save_mat(&C2, mat_a_col, mat_b_row);
	//free memory
	free(A); 
	free(B);
	free(C1);
	free(C2);
	exit(res);
}

