#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <fstream>
const int N = 16;
const int grid_x_size = 5000;
const int grid_y_size = 5000;
const int block_x_size = 1;
const int block_y_size = 1;
const int mat_size = 5000;
using namespace std;
__global__ void gpuMatMultKernel(const float *a, const float *b, float *result)
{
	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x
		+ blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < mat_size*mat_size)
	{
		int row = threadId / mat_size;
		int column = threadId % mat_size;

		result[threadId] = 0;
		for (int i = 0; i < N; i++)
		{
			result[threadId] += a[row * mat_size + i] * b[i * mat_size + column];
		}
	}
}

__global__ void hello(char *a, int *b)
{
	printf("hello %d %d %d \n", threadIdx.x, b[threadIdx.x], a[0]);
	a[threadIdx.x] += b[threadIdx.x];
	printf("hello %d %d %d \n", threadIdx.x, b[threadIdx.x], a[0]);
}

void print_mat(float* src) {
	for (int i = 0; i < mat_size; i++)
	{
		for (int j = 0; j < mat_size; j++)
		{
			printf("%f ", src[i*mat_size + j]);
		}
		printf("\n");
	}
}
void save_mat(float* src) {
	ofstream outfile("res.txt");
	for (int i = 0; i < mat_size; i++)
	{
		for (int j = 0; j < mat_size; j++)
		{
			outfile << i << " " << j << " " << src[i*mat_size + j] << endl;
		}
	}
	outfile.close();
}
int main()
{
	float* mat_a = (float*)malloc(sizeof(float)*mat_size*mat_size);
	float* mat_b = (float*)malloc(sizeof(float)*mat_size*mat_size);
	float* mat_c = (float*)malloc(sizeof(float)*mat_size*mat_size);
	for (int i = 0; i < mat_size; i++)
	{
		for (int j = 0; j < mat_size; j++)
		{
			mat_a[i*mat_size + j] = i - 0.1*j + 1;
			mat_b[i*mat_size + j] = 0.2*j - 0.1*i;
		}
	}
	//print_mat(mat_a);
	//print_mat(mat_b);
	float* mat_a_cuda;
	float* mat_b_cuda;
	float* mat_c_cuda;
	hipMalloc((void**)&mat_a_cuda, sizeof(float)*mat_size*mat_size);
	hipMalloc((void**)&mat_b_cuda, sizeof(float)*mat_size*mat_size);
	hipMalloc((void**)&mat_c_cuda, sizeof(float)*mat_size*mat_size);
	hipMemcpy(mat_a_cuda, mat_a, sizeof(float)*mat_size*mat_size, hipMemcpyHostToDevice);
	hipMemcpy(mat_b_cuda, mat_b, sizeof(float)*mat_size*mat_size, hipMemcpyHostToDevice);
	dim3 dimBlock(block_x_size, block_y_size);
	dim3 dimGrid(grid_x_size, grid_y_size);
	gpuMatMultKernel <<<dimGrid, dimBlock >> > (mat_a_cuda, mat_b_cuda,mat_c_cuda);
	printf("finish\n");
	hipMemcpy(mat_c, mat_c_cuda, sizeof(float)*mat_size*mat_size, hipMemcpyDeviceToHost);
	//print_mat(mat_c);
	hipFree(mat_a_cuda);
	hipFree(mat_b_cuda);
	save_mat(mat_c);
	return 0;
}
