#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <fstream>

#include <stdio.h>
#include <stdlib.h>
#include <fstream>
using namespace std;

__global__ void hello()
{
	printf("hello from block %d %d thread %d  %d \n", blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
}

int main()
{
	dim3 dimBlock(8, 16);
	dim3 dimGrid(2, 4);
	hello << <dimGrid, dimBlock >> > ();
	return 0;
}